#include "hip/hip_runtime.h"
#include "./gather_point-inl.h"
#include <mshadow/tensor.h>

namespace mxnet {
namespace cuda {

__global__ void gatherpointKernel(int b,int n,int m
    ,const float * __restrict__ inp,const int * __restrict__ idx, float * __restrict__ out){
    for (int i=blockIdx.x;i<b;i+=gridDim.x){
		for (int j=blockIdx.y*blockDim.x+threadIdx.x;j<m;j+=blockDim.x*gridDim.y){
			int a=idx[i*m+j];
			out[(i*m+j)*3+0]=inp[(i*n+a)*3+0];
			out[(i*m+j)*3+1]=inp[(i*n+a)*3+1];
			out[(i*m+j)*3+2]=inp[(i*n+a)*3+2];
		}
    }
}

__global__ void scatteraddpointKernel(int b,int n,int m
    ,const float * __restrict__ out_g,const int * __restrict__ idx, float * __restrict__ inp_g){
    for (int i=blockIdx.x;i<b;i+=gridDim.x){
		for (int j=blockIdx.y*blockDim.x+threadIdx.x;j<m;j+=blockDim.x*gridDim.y){
			int a=idx[i*m+j];
			atomicAdd(&inp_g[(i*n+a)*3+0],out_g[(i*m+j)*3+0]);
			atomicAdd(&inp_g[(i*n+a)*3+1],out_g[(i*m+j)*3+1]);
			atomicAdd(&inp_g[(i*n+a)*3+2],out_g[(i*m+j)*3+2]);
		}
    }
}

} // namespace cuda

namespace op {

template<>
void GatherPointCompute<gpu>(const nnvm::NodeAttrs& attrs,
                        const OpContext& ctx,
						const std::vector<TBlob>& inputs,
                        const std::vector<OpReqType>& req,
                        const std::vector<TBlob>& outputs) {
	const int B = inputs[0].size(0);
	const int N = inputs[0].size(1);
	const int M = inputs[1].size(1);
	auto *stream = ctx.get_stream<gpu>();
	auto s = mshadow::Stream<mxnet::gpu>::GetStream(stream);
	dim3 grid_dim = dim3(2, 8, 1);
	cuda::gatherpointKernel<<<grid_dim, 512, 0, s>>>(
		B, N, M, inputs[0].dptr<float>(), inputs[1].dptr<int>(), outputs[0].dptr<float>());
}

template<>
void GatherPointGradCompute<gpu>(const nnvm::NodeAttrs& attrs,
                            const OpContext& ctx, 
							const std::vector<TBlob>& inputs,
                            const std::vector<OpReqType>& req,
                            const std::vector<TBlob>& outputs) {
	const TBlob &out_grad = inputs[0];
	const TBlob &idx = inputs[1];
	const TBlob &in_grad = outputs[0];

	const int B = in_grad.size(0);
	const int N = in_grad.size(1);
	const int M = idx.size(1);
	auto *stream = ctx.get_stream<gpu>();
	auto s = mshadow::Stream<mxnet::gpu>::GetStream(stream);
	dim3 grid_dim = dim3(2, 8, 1);
	cuda::scatteraddpointKernel<<<grid_dim, 512, 0, s>>>(
		B, N, M, out_grad.dptr<float>(), idx.dptr<int>(), in_grad.dptr<float>());
}

NNVM_REGISTER_OP(_contrib_GatherPoint)
.set_attr<FCompute>("FCompute<gpu>", GatherPointCompute<gpu>);

NNVM_REGISTER_OP(_backward_contrib_GatherPoint)
.set_attr<FCompute>("FCompute<gpu>", GatherPointGradCompute<gpu>);

} // namespace op
} // namespace mxnet
