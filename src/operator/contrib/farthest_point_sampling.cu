#include "hip/hip_runtime.h"
#include "./farthest_point_sampling-inl.h"
#include <mshadow/tensor.h>

namespace mxnet {
namespace cuda {

    __global__ void farthestpointsamplingKernel(int b,int n,int m
        ,const float * __restrict__ dataset,float * __restrict__ temp,int * __restrict__ idxs){
      if (m<=0)
        return;
      const int BlockSize=512;
      __shared__ float dists[BlockSize];
      __shared__ int dists_i[BlockSize];
      const int BufferSize=3072;
      __shared__ float buf[BufferSize*3];
      for (int i=blockIdx.x;i<b;i+=gridDim.x){
        int old=0;
        if (threadIdx.x==0)
          idxs[i*m+0]=old;
        for (int j=threadIdx.x;j<n;j+=blockDim.x){
          temp[blockIdx.x*n+j]=1e38;
        }
        for (int j=threadIdx.x;j<min(BufferSize,n)*3;j+=blockDim.x){
          buf[j]=dataset[i*n*3+j];
        }
        __syncthreads();
        for (int j=1;j<m;j++){
          int besti=0;
          float best=-1;
          float x1=dataset[i*n*3+old*3+0];
          float y1=dataset[i*n*3+old*3+1];
          float z1=dataset[i*n*3+old*3+2];
          for (int k=threadIdx.x;k<n;k+=blockDim.x){
            float td=temp[blockIdx.x*n+k];
            float x2,y2,z2;
            if (k<BufferSize){
              x2=buf[k*3+0];
              y2=buf[k*3+1];
              z2=buf[k*3+2];
            }else{
              x2=dataset[i*n*3+k*3+0];
              y2=dataset[i*n*3+k*3+1];
              z2=dataset[i*n*3+k*3+2];
            }
            float d=(x2-x1)*(x2-x1)+(y2-y1)*(y2-y1)+(z2-z1)*(z2-z1);
            float d2=min(d,td);
            if (d2!=td)
              temp[blockIdx.x*n+k]=d2;
            if (d2>best){
              best=d2;
              besti=k;
            }
          }
          dists[threadIdx.x]=best;
          dists_i[threadIdx.x]=besti;
          for (int u=0;(1<<u)<blockDim.x;u++){
            __syncthreads();
            if (threadIdx.x<(blockDim.x>>(u+1))){
              int i1=(threadIdx.x*2)<<u;
              int i2=(threadIdx.x*2+1)<<u;
              if (dists[i1]<dists[i2]){
                dists[i1]=dists[i2];
                dists_i[i1]=dists_i[i2];
              }
            }
          }
          __syncthreads();
          old=dists_i[0];
          if (threadIdx.x==0)
            idxs[i*m+j]=old;
        }
      }
    }

} // namespace cuda

namespace op {

template<>
void FarthestPointSamplingCompute<gpu>(const nnvm::NodeAttrs& attrs,
                        const OpContext& ctx,
						const std::vector<TBlob>& inputs,
                        const std::vector<OpReqType>& req,
                        const std::vector<TBlob>& outputs) {
using namespace mshadow;
CHECK_GT(ctx.requested.size(), 0);
	const int B = inputs[0].size(0);
	const int N = inputs[0].size(1);
    const FarthestPointSamplingParam& param = nnvm::get<FarthestPointSamplingParam>(attrs.parsed);

	auto *stream = ctx.get_stream<gpu>();
	auto s = mshadow::Stream<mxnet::gpu>::GetStream(stream);
    auto tmp_shape = mshadow::Shape2(32, N);

    Tensor<gpu, 2, float> tmp = ctx.requested[0].get_space_typed<gpu, 2, float>(tmp_shape, stream);
    // Fill<false>(stream, TBlob(reinterpret_cast<nnvm::dim_t*>(tmp.dptr_), tmp_shape, gpu::kDevMask), kWriteTo, 1e 10);

	cuda::farthestpointsamplingKernel<<<32, 512, 0, s>>>(
		B, N, param.npoints, inputs[0].dptr<float>(), tmp.dptr_, outputs[0].dptr<int>());
}

template<>
void FarthestPointSamplingGradCompute<gpu>(const nnvm::NodeAttrs& attrs,
                            const OpContext& ctx, 
							const std::vector<TBlob>& inputs,
                            const std::vector<OpReqType>& req,
                            const std::vector<TBlob>& outputs) {

}

NNVM_REGISTER_OP(_contrib_FarthestPointSampling)
.set_attr<FCompute>("FCompute<gpu>", FarthestPointSamplingCompute<gpu>);

NNVM_REGISTER_OP(_backward_contrib_FarthestPointSampling)
.set_attr<FCompute>("FCompute<gpu>", FarthestPointSamplingGradCompute<gpu>);

} // namespace op
} // namespace mxnet
